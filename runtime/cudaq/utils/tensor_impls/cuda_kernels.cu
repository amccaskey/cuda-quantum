#include "hip/hip_runtime.h"
// cuda_tensor_kernels.cu
#include "hip/hip_complex.h"
#include "cuda_kernels.h"

__device__ __host__ hipDoubleComplex operator*(hipDoubleComplex a,
                                              hipDoubleComplex b) {
  return hipCmul(a, b);
}
__device__ __host__ hipDoubleComplex operator+(hipDoubleComplex a,
                                              hipDoubleComplex b) {
  return hipCadd(a, b);
}
__device__ __host__ hipFloatComplex operator*(hipFloatComplex a,
                                             hipFloatComplex b) {
  return hipCmulf(a, b);
}
__device__ __host__ hipFloatComplex operator+(hipFloatComplex a,
                                             hipFloatComplex b) {
  return hipCaddf(a, b);
}

__device__ inline hipDoubleComplex operator%(const hipDoubleComplex &a,
                                            const hipDoubleComplex &b) {
  double real = fmod(hipCreal(a), hipCreal(b));
  double imag = fmod(hipCimag(a), hipCimag(b));
  return make_hipDoubleComplex(real, imag);
}

__device__ inline hipFloatComplex operator%(const hipFloatComplex &a,
                                           const hipFloatComplex &b) {
  float real = fmodf(hipCrealf(a), hipCrealf(b));
  float imag = fmodf(hipCimagf(a), hipCimagf(b));
  return make_hipFloatComplex(real, imag);
}

namespace cudaq {
namespace kernels {

template <typename Scalar>
__global__ void elementwise_add_kernel(const Scalar *a, const Scalar *b,
                                       Scalar *c, size_t n) {
  const int tid = blockDim.x * blockIdx.x + threadIdx.x;
  if (tid < n) {
    c[tid] = a[tid] + b[tid];
  }
}

template <typename Scalar>
void elementwise_add_kernel(int threads, int blocks, const Scalar *a,
                            const Scalar *b, Scalar *c, size_t n) {
  kernels::elementwise_add_kernel<<<blocks, threads>>>(a, b, c, n);
}

template <typename Scalar>
__global__ void elementwise_multiply_kernel(const Scalar *a, const Scalar *b,
                                            Scalar *c, size_t n) {
  const int tid = blockDim.x * blockIdx.x + threadIdx.x;
  if (tid < n) {
    c[tid] = a[tid] * b[tid];
  }
}

template <typename Scalar>
void elementwise_multiply_kernel(int threads, int blocks, const Scalar *a,
                                 const Scalar *b, Scalar *c, size_t n) {
  elementwise_multiply_kernel<<<blocks, threads>>>(a, b, c, n);
}

// template <typename Scalar>
// __global__ void slice_kernel(const Scalar* input, Scalar* output,
//                            const size_t* input_shape, const size_t*
//                            output_shape, const slice* slices, const int rank)
//                            {
//     // Get global thread ID
//     const int tid = blockDim.x * blockIdx.x + threadIdx.x;

//     // Calculate total output size
//     size_t output_size = 1;
//     for (int i = 0; i < rank; i++) {
//         output_size *= output_shape[i];
//     }

//     if (tid < output_size) {
//         // Convert linear index to multi-dimensional indices
//         size_t remaining = tid;
//         size_t input_index = 0;
//         size_t stride = 1;

//         for (int i = rank - 1; i >= 0; i--) {
//             const size_t output_idx = remaining % output_shape[i];
//             remaining /= output_shape[i];

//             // Calculate input index using slice information
//             const size_t start = slices[i].start ? *slices[i].start : 0;
//             const size_t step = slices[i].step ? *slices[i].step : 1;
//             const size_t input_idx = start + output_idx * step;

//             input_index += input_idx * stride;
//             stride *= input_shape[i];
//         }

//         output[tid] = input[input_index];
//     }
// }

template <typename Scalar>
__global__ void elementwise_modulo_kernel(const Scalar *a, const Scalar *b,
                                          Scalar *c, size_t n) {
  const int tid = blockDim.x * blockIdx.x + threadIdx.x;
  if (tid < n) {
    c[tid] = a[tid] % b[tid];
  }
}

template <typename Scalar>
void elementwise_modulo_kernel(int threads, int blocks, const Scalar *a,
                               const Scalar *b, Scalar *c, size_t n) {
  elementwise_modulo_kernel<<<blocks, threads>>>(a, b, c, n);
}

template <typename Scalar>
__global__ void scalar_modulo_kernel(const Scalar *a, Scalar value, Scalar *c,
                                     size_t n) {
  const int tid = blockDim.x * blockIdx.x + threadIdx.x;
  if (tid < n) {
    c[tid] = a[tid] % value;
  }
}
template <typename Scalar>
void scalar_modulo_kernel(int threads, int blocks, const Scalar *a,
                          Scalar value, Scalar *c, size_t n) {
  scalar_modulo_kernel<<<blocks, threads>>>(a, value, c, n);
}

// Explicit template instantiations
#define INSTANTIATE_KERNELS(TYPE)                                              \
  template __global__ void elementwise_add_kernel<TYPE>(                       \
      const TYPE *, const TYPE *, TYPE *, size_t);                             \
  template void elementwise_add_kernel<TYPE>(int, int, const TYPE *,           \
                                             const TYPE *, TYPE *, size_t);    \
  template __global__ void elementwise_multiply_kernel<TYPE>(                  \
      const TYPE *, const TYPE *, TYPE *, size_t);                             \
  template void elementwise_multiply_kernel<TYPE>(                             \
      int, int, const TYPE *, const TYPE *, TYPE *, size_t);                   \
  template __global__ void elementwise_modulo_kernel<TYPE>(                    \
      const TYPE *, const TYPE *, TYPE *, size_t);                             \
  template void elementwise_modulo_kernel<TYPE>(int, int, const TYPE *,        \
                                                const TYPE *, TYPE *, size_t); \
  template __global__ void scalar_modulo_kernel<TYPE>(const TYPE *, TYPE,      \
                                                      TYPE *, size_t);         \
  template void scalar_modulo_kernel<TYPE>(int, int, const TYPE *, TYPE,       \
                                           TYPE *, size_t);

INSTANTIATE_KERNELS(int)
INSTANTIATE_KERNELS(uint8_t)
INSTANTIATE_KERNELS(float)
INSTANTIATE_KERNELS(double)
INSTANTIATE_KERNELS(hipFloatComplex)
INSTANTIATE_KERNELS(hipDoubleComplex)
INSTANTIATE_KERNELS(size_t)

#undef INSTANTIATE_KERNELS

} // namespace kernels
} // namespace cudaq
